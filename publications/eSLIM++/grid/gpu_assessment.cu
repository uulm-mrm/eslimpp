#include "hip/hip_runtime.h"
#include "functions.hpp"
#include <iostream>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(ans)                                                                                                \
  do                                                                                                                   \
  {                                                                                                                    \
    if ((ans) != hipSuccess)                                                                                          \
    {                                                                                                                  \
      printf("CUDA Error: %s \nFile: %s \nLine: %d\n", hipGetErrorString((ans)), __FILE__, __LINE__);                 \
    }                                                                                                                  \
  } while (0)

__global__ void run_fusion(Opinion* a, Opinion* b, Opinion* dest, int* classes,  std::size_t size) {
  std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size) {
    return;
  }
  // a[idx] = Opinion(0.0,0.8);
  // b[idx] = Opinion(0.8,0.0);

  dest[idx] = a[idx].cum_fuse(b[idx]);
  if (dest[idx].uncertainty() > 0.5) {
    classes[idx] = 0;
  }
  else {
    typename Opinion::FLOAT_t prob = dest[idx].getBinomialProjection();
    if (prob > 0.7) {
      classes[idx] = 1;
    }
    else if (prob < 0.3) {
      classes[idx] = 2;
    }
    else {
      classes[idx] = 3;
    }
  }
}

TimeDiffs run_gpu_assessment(const std::size_t n_ops, const std::size_t n_runs, const std::vector<Opinion>& sensor_a, const std::vector<Opinion>& sensor_b)
{
  TimeDiffs runtimes(n_runs);
  std::size_t map_size_byte = n_ops * sizeof(Opinion);
  std::cout << "size of single subjective_logic_lib::OpinionNoBase<2,float>: " << sizeof(Opinion) << std::endl;
  Opinion* a{nullptr};
  Opinion* b{nullptr};
  Opinion* dest{nullptr};
  int* classes{nullptr};

  constexpr std::size_t blocks = 512;
  auto threads_per_block = static_cast<std::size_t>(std::ceil(static_cast<double>(n_ops) / blocks));

  hipMalloc((void**)&a, map_size_byte);
  hipMalloc((void**)&b, map_size_byte);
  hipMalloc((void**)&dest, map_size_byte);
  hipMalloc((void**)&classes, map_size_byte);

  hipMemcpy(a, sensor_a.data(), map_size_byte, hipMemcpyHostToDevice);
  hipMemcpy(b, sensor_b.data(), map_size_byte, hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  // warm up
  run_fusion<<<threads_per_block, blocks>>>(a,b,dest, classes, n_ops);

  hipEvent_t cu_start, cu_stop;
  hipEventCreate(&cu_start);
  hipEventCreate(&cu_stop);

  for (std::size_t i{0}; i< n_runs; ++i) {
    hipEventRecord(cu_start);
    run_fusion<<<threads_per_block, blocks>>>(a,b,dest, classes, n_ops);
    hipEventRecord(cu_stop);
    CHECK_CUDA(hipPeekAtLastError());
    CHECK_CUDA(hipDeviceSynchronize());
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, cu_start, cu_stop);
    runtimes[i] = TimeDiff{static_cast<std::uint64_t>(milliseconds*1e6)};
  }
  CHECK_CUDA(hipPeekAtLastError());
  CHECK_CUDA(hipDeviceSynchronize());

  hipEventDestroy(cu_start);
  hipEventDestroy(cu_stop);

  std::vector<int> results(n_ops);
  hipMemcpy(results.data(), classes, n_ops * sizeof(int), hipMemcpyDeviceToHost);

  std::vector<int> hist;
  hist.resize(4);
  for (const int entry : results) {
    hist[entry] += 1;
  }
  double denom = hist[1] + hist[3];
  double score = hist[3] / denom;
  std::cout << "size of one map with " << n_ops << " elements is: " << map_size_byte / 1e6 << "MB" << std::endl;
  std::cout << "the self-assessment score is: " << score << std::endl;

  hipFree(a);
  hipFree(b);
  hipFree(dest);

  return runtimes;
}
